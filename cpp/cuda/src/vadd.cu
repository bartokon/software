
#include <hip/hip_runtime.h>
__global__ void vadd(int *a, int *b, int *c, int N){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        c[index] = a[index] + b[index];
    }
};

void vadd_wrapper(int *a, int *b, int *c, int samples) {
    int *a_gpu, *b_gpu, *c_gpu;
    hipMalloc(&a_gpu, sizeof(int)*samples);
    hipMalloc(&b_gpu, sizeof(int)*samples);
    hipMalloc(&c_gpu, sizeof(int)*samples);
    hipMemcpy(a_gpu, a, sizeof(int)*samples, hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, sizeof(int)*samples, hipMemcpyHostToDevice);
    vadd<<<1, samples>>>(a_gpu, b_gpu, c_gpu, samples);
    hipMemcpy(c, c_gpu, sizeof(int)*samples, hipMemcpyDeviceToHost);
    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);
};
